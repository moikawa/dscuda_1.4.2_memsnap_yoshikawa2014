#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <cutil.h>
#include <cutil_inline.h>


//16からはじめて倍々
//プロットはlogscale
#define N (1024)


__global__ void matrixMul(float *A, float *B, float *C, int size) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < size && col < size) {
    float x = 0.0f;
    for (int k = 0; k < size; k++) {
      x += A[row * size + k] * B[k * size + col];
    }

    C[row * size + col] = x;
  }
}

void randomInit(float *x, int size, float max = 1.0f) {
  for (int i = 0; i < size; i++) {
    x[i] = max * (rand() / (float)RAND_MAX);
  }
}

int main(void) {
  float *hA, *hB, *hC;
  float *dA, *dB, *dC;
  dim3 block(16, 16);
  dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
  struct timeval start, end;

  hA = new float[N * N];
  hB = new float[N * N];
  hC = new float[N * N];

  randomInit(hA, N * N, 10);
  randomInit(hB, N * N, 10);


  int size = N * N * sizeof(float);

  gettimeofday(&start, NULL);
  hipMalloc((void**)&dA, size);
  hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);

  hipMalloc((void**)&dB, size);
  hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

  hipMalloc((void**)&dC, size);


  matrixMul<<<grid, block>>>(dA, dB, dC, N);

  hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);
  gettimeofday(&end, NULL);

  printf("%f\n", (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0);


  hipFree(dA);
  hipFree(dB);
  hipFree(dC);

  delete [] hA;
  delete [] hB;
  delete [] hC;
  
  return 0;
}
